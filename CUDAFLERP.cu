#include "hip/hip_runtime.h"
#include "CUDAFLERP.h"

__global__ void CUDAFLERP_kernel(const hipTextureObject_t d_img_tex, const float gxs, const float gys, float* __restrict const d_out, const int neww) {
	uint32_t x = (blockIdx.x << 9) + (threadIdx.x << 1);
	const uint32_t y = blockIdx.y;
	const float fy = (y + 0.5f)*gys - 0.5f;
	const float wt_y = fy - floor(fy);
	const float invwt_y = 1.0f - wt_y;
#pragma unroll
	for (int i = 0; i < 2; ++i, ++x) {
		const float fx = (x + 0.5f)*gxs - 0.5f;
		const float4 f = tex2Dgather<float4>(d_img_tex, fx + 0.5f, fy + 0.5f);
		const float wt_x = fx - floor(fx);
		const float invwt_x = 1.0f - wt_x;
		const float xa = invwt_x*f.w + wt_x*f.z;
		const float xb = invwt_x*f.x + wt_x*f.y;
		const float res = invwt_y*xa + wt_y*xb;
		if (x < neww) d_out[y*neww + x] = res;
	}
}

void CUDAFLERP(const hipTextureObject_t d_img_tex, const int oldw, const int oldh, float* __restrict const d_out, const uint32_t neww, const uint32_t newh) {
	const float gxs = static_cast<float>(oldw) / static_cast<float>(neww);
	const float gys = static_cast<float>(oldh) / static_cast<float>(newh);
	CUDAFLERP_kernel<<<{((neww - 1) >> 9) + 1, newh}, 256>>>(d_img_tex, gxs, gys, d_out, neww);
	hipDeviceSynchronize();
}